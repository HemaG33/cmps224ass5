#include "hip/hip_runtime.h"

#include "common.h"

#include "timer.h"

__global__ void histogram_private_kernel(unsigned char* image, unsigned int* bins, unsigned int width, unsigned int height) {

    __shared__ unsigned int private_bins[256];

    // Initialize private histogram bins to zero
    for (int i = 0; i < 256; ++i) {
        private_bins[i] = 0;
    }

    __syncthreads(); 

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width * height) {
        unsigned char b = image[i];
        atomicAdd(&private_bins[b], 1);
    }

    __syncthreads();

    // Commit non-zero bin counts to the global copy of the histogram
    if (threadIdx.x < 256) {
        atomicAdd(&bins[threadIdx.x], private_bins[threadIdx.x]);
    }




}

void histogram_gpu_private(unsigned char* image_d, unsigned int* bins_d, unsigned int width, unsigned int height) {

    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (width * height + numThreadsPerBlock - 1) / numThreadsPerBlock;
    histogram_private_kernel<<<numBlocks, numThreadsPerBlock>>>(image_d, bins_d, width, height);

}
__global__ void histogram_private_coarse_kernel(unsigned char* image, unsigned int* bins, unsigned int width, unsigned int height) {
    __shared__ unsigned int private_bins[256]; // Private histogram bins for each block

    // Initialize private histogram bins to zero
    for (int j = 0; j < 256; j++) {
        private_bins[j] = 0;
    }

    __syncthreads(); // Ensure all threads have initialized private histogram bins

    unsigned int coarsening_factor = 64; // Coarsening factor variable

    unsigned int tid = blockIdx.x * (blockDim.x * coarsening_factor) + threadIdx.x; // Adjusted tid calculation for coarsening factor
    for (int i = 0; i < coarsening_factor; ++i) { // Iterate over pixels based on the coarsening factor
        unsigned int index = tid + i * blockDim.x;
        if (index < width * height) {
            unsigned char b = image[index];
            atomicAdd(&private_bins[b], 1); // Update private histogram bins atomically
        }
    }

    __syncthreads(); // Ensure all threads have finished updating private histogram bins

    // Commit non-zero bin counts to the global copy of the histogram in parallel
    if (threadIdx.x < 256) {
        atomicAdd(&bins[threadIdx.x], private_bins[threadIdx.x]);
    }
}

void histogram_gpu_private_coarse(unsigned char* image_d, unsigned int* bins_d, unsigned int width, unsigned int height) {
    unsigned int coarsening_factor = 64; // Define the coarsening factor


    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (width * height + numThreadsPerBlock * coarsening_factor - 1) / (numThreadsPerBlock * coarsening_factor);
    histogram_private_coarse_kernel<<<numBlocks, numThreadsPerBlock>>>(image_d, bins_d, width, height);
}

